
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime_api.h>

#define SQ(x) ((x) * (x))

static const float A = -4.0, B = 4.0;  // limites de integración
static const int N = 1 << 22;          // número de intervalos = 2^22
static const float H = (B - A) / N;    // tamaño del intervalo de integración
static const float PI(M_PI);           // π con precision simple

__device__ float h(float x) {
    return .5f + 1.5f / (1.0f + 50.0f * SQ(x));
}

float host_h(float x) {
    return .5f + 1.5f / (1.0f + 50.0f * SQ(x));
}

__device__ float f(float x) {
    int i;
    float sum = 0.0f, x0;
    for (i = 0; i < 10; ++i){
        x0 = -3.3f + i * 0.7f;
        sum += h(x - x0);
    }
    return sum/10.0f;
}

float host_f(float x) {
    int i;
    float sum = 0.0f, x0;
    for (i = 0; i < 10; ++i){
        x0 = -3.3f + i * 0.7f;
        sum += host_h(x - x0);
   }
    return sum/10.0f;
}

__device__ float g(float x) {
    float c = cosf(2.0f * PI * f(x) * x);
    return expf(-x/16.0f) * SQ(c);
}


float host_g(float x) {
    float c = cosf(2.0f * PI * host_f(x) * x);
    return expf(-x/16.0f) * SQ(c);
}

 __global__ void
clean_blocks(float subtotals[]){
    subtotals[blockIdx.x] = 0;
}

__global__ void
integrate_blocks(float subtotals[]) {
	// Inicializar variable __shared__ 
	__shared__ float partialValues[512];
	//extern __shared__ float partialValues[];

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int tx = threadIdx.x;

	// Cada thread según su id, calcula la función de g(x)
	float x = 0; 

	x = A + i * (H / 2);
	partialValues[tx] = (i % 2 == 0 ? 4 : 2) * g(x);


	 // Sincronizamos las hebras una vez que termine
	__syncthreads(); 

	// Hacer una suma por reduccion con los valores del arreglo subtotals[]
	for (int offset = blockDim.x/2; offset > 0; offset >>=1){
	        if (tx < offset)
	        {
	            //Añadimos una suma parcial con el offset
	            partialValues[tx] += partialValues[tx+offset];
	        }
	        __syncthreads();
	}
        // Escribimos el resultado del primer elemento de nuestro arreglo
        if(tx == 0)
        {
        // Resultados lo guarda por bloques y no por hebras
        subtotals[blockIdx.x] = partialValues[0];
        }
}

__global__ void
reduction(float subtotals[]) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int tx = threadIdx.x;


	for (int offset = blockDim.x/2; offset > 0; offset >>=1){
	        if (tx < offset)
	        {
	            subtotals[i] += subtotals[i+offset];
	        }
	        __syncthreads();
	}
        if(tx == 0)
        {
	        subtotals[blockIdx.x] = subtotals[0];
        }
}

__global__ void
final_reduction(float subtotals[]) {
    float suma;
    int n = 2 * N / blockDim.x;

    if (threadIdx.x == 0){
        for (int i = 0; i < n ; i+= blockDim.x){
            suma += subtotals[i];
	}
        suma += g(B) - g(A);
        subtotals[0] = suma;
    }

}

int main(int argc, char *argv[]) {

    // El programa recibe como parámetro el número de hebras por bloque.
    // Recuerden que este número debe ser múltiplo de 32 (tamaño del warp)
    // y puede ser a lo más 512 (limitación del hardware).

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " threads_per_block" << std::endl;
        std::exit(1);
    }
    int block_size = std::atoi(argv[1]);


    // Al usar N subintervalos, hay que evaluar la función en 2N + 1 puntos.
    // Para paralelizar, mejor evaluar 2N puntos y sumar el último al final.
    // Por lo tanto, necesitamos 2N hebras.
    int nr_blocks = 2 * N / block_size;
    size_t sharedMem = 512;

    // Reservar arreglos en RAM y en la GPU para guardar los resultados.
    float *subtotals_h, *subtotals_d;
    subtotals_h = new float[nr_blocks];
    hipMalloc((void **) &subtotals_d, sizeof(float) * nr_blocks);

    // kernel para limpiar el valor del arreglo
    clean_blocks<<<nr_blocks, 1>>>(subtotals_d);

    // kernel para calcular los valores de la suma de la integral
    integrate_blocks<<<nr_blocks, block_size,sharedMem>>>(subtotals_d);

    // kernel para hacer la primera reduccion a solo un bloque
    reduction<<<nr_blocks/block_size, block_size>>>(subtotals_d);

    // kernel para realizar la ultima reduccion en un bloque y obtener la suma total.
    final_reduction<<<1,block_size>>>(subtotals_d);

    hipMemcpy(subtotals_h, subtotals_d, sizeof(float) * nr_blocks, hipMemcpyDeviceToHost);

    float sum = subtotals_h[0];

    float integral = sum * H / 6.0f;

    std::cout << "Integral: " << std::setprecision(5) << integral << std::endl;
    sum = 0.0;
    hipFree(subtotals_d);
    std::free(subtotals_h);
}

